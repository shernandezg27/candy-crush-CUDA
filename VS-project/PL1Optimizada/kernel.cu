#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <queue>
#include <cstdlib>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <string.h>
using namespace std;

//CONTROL DE ERRORES
int colores = 6;
int vidas = 5;
__constant__ int dev_pos[2];


__device__ bool esta_conectado(int posInicial, int posObjetivo, int* matriz, int dev_M, int dev_N)
{
    if (posInicial == posObjetivo) return true;
    if (matriz[posInicial] != matriz[posObjetivo]) return false;

    int mov_adyacentes[4] = { 1, -1, dev_N, -dev_N }; // Array de posibles movimientos que se pueden hacer para buscar casillas contiguas
    int* visitados = new int[dev_M * dev_N]; // Array para guardar las posiciones ya visitadas
    for (int i = 0; i < dev_M * dev_N; i++) {
        visitados[i] = 0;
    }
    int* visitados_temp = new int[dev_M * dev_N];  // Array temporal para marcar visitados en la siguiente iteraci�n
    for (int i = 0; i < dev_M * dev_N; i++) {
        visitados_temp[i] = 0;
    }
    int* posiciones_por_visitar = new int[dev_M * dev_N]; // Array para guardar las posiciones a visitar en la siguiente iteraci�n 
    int num_posiciones_por_visitar = 0; // N�mero de posiciones a visitar en la siguiente iteraci�n

    visitados[posInicial] = 1;
    visitados_temp[posInicial] = 1;
    posiciones_por_visitar[num_posiciones_por_visitar++] = posInicial;

    while (num_posiciones_por_visitar > 0) {
        int pos_actual = posiciones_por_visitar[--num_posiciones_por_visitar];

        for (int i = 0; i < 4; i++) {
            int adyacente = pos_actual + mov_adyacentes[i];

            if (adyacente >= 0 && adyacente < dev_M * dev_N) { //si no nos hemos salido de la matriz       
                if (adyacente / dev_N == pos_actual / dev_N || adyacente % dev_N == pos_actual % dev_N) { //si no hemos saltado de linea con un desplazamiento horizontal (se comprueba si esta en la misma fila o la misma columna)
                    if (visitados[adyacente] != 1) { // si aun no hemos pasado por esa posici�n
                        if (matriz[posObjetivo] == matriz[adyacente]) {
                            if (adyacente == posObjetivo) {
                                return true;
                            }
                            else {
                                visitados[adyacente] = 1;
                                visitados_temp[adyacente] = 1;
                                posiciones_por_visitar[num_posiciones_por_visitar++] = adyacente;
                            }
                        }
                    }
                }
            }
        }

        // Marcar como visitados las posiciones de la iteraci�n actual
        for (int i = 0; i < dev_M * dev_N; i++) {
            visitados[i] = visitados[i] | visitados_temp[i];
            visitados_temp[i] = 0;
        }
    }

    return false;
}


__device__ int rellenar(int posInicial, int* matriz, unsigned long long seed, int colores, int dev_M, int dev_N)
{
    hiprandState state;
    hiprand_init(seed, threadIdx.x, 0, &state);

    int primerCero = (posInicial % dev_N) + (dev_M * (dev_N - 1)); //Nos colocamos en la posici�n m�s baja de la columna de la posicion que queremos rellenar
    while (matriz[primerCero] != 0 && primerCero >= 0) {
        primerCero = primerCero - dev_N;
    }   //con esto tenemos la posici�n del primer cero que se encuentra en la columna
    if (primerCero < 0 || primerCero < posInicial) { //si no hay ceros o estan por encima de la posicion que se quiere rellenar se deja como estaba
        return matriz[posInicial];
    }
    int pos_desde_primer_cero = ((primerCero - posInicial) / dev_N) + 1;  //posiciones en vertical desde el primer cero hasta la posicion que se quiere rellenar
    //printf("%d ", pos_desde_primer_cero);
    int resultado = 0;
    int posActual = primerCero;
    for (int i = 0; i < pos_desde_primer_cero; i++) {   //buscamos el numero en la posicion pos_desde_primer_cero sin contar los ceros, que ser� la que le corresponda a la posici�n que queremos al bajar todos los bloques
        while (posActual >= 0 && matriz[posActual] == 0) {    //saltamos los ceros o si nos hemos salido de la matriz
            posActual = posActual - dev_N;
        }
        if (posActual < 0) {
            int aleatorio = hiprand(&state) % colores + 1;
            return aleatorio;
        }
        resultado = matriz[posActual];
        posActual = posActual - dev_N;
    }
    return resultado;
}



__global__ void addKernel(int* dev_tablero, int* dev_resultado, int* dev_bloques_eliminados, int fila_o_columna, unsigned long long seed, int colores, int dev_M, int dev_N)
{
    //Calculamos la posici�n en la que va a operar el hilo
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Si se encuentra dentro de los limites de la matriz hace los calculos
    if (i < dev_M && j < dev_N) {
        int pos_hilo = i * dev_N + j;
        int pos_seleccionada = dev_pos[0] * dev_N + dev_pos[1];
        bool borrar = false;
        int valor_propio_inicial = dev_tablero[pos_hilo]; //ayuda a reducir los accesos a memoria global
        extern __shared__ int shared_tablero[];
        shared_tablero[threadIdx.x * blockDim.y + threadIdx.y] = valor_propio_inicial;
        __syncthreads();

        //Buscar que bloques se van a eliminar
        int valor_pos_seleccionada;
        int i_seleccionada = dev_pos[0];
        int j_seleccionada = dev_pos[1];
        /*
        * queremos saber si el bloque seleccionado se encuentra en el mismo bloque de hilos que el que ejecuta esto para saber si aceder a memoria compartida o global
        * i_seleccionada >= blockIdx.x * blockDim.x; comprueba que esta por debajo de la fila superior del bloque
        * i_seleccionada < blockIdx.x * blockDim.x + blockDim.x; comprueba que esta por encima de la fila inferior del bloque
        * 
        * j_seleccionada >= blockIdx.y * blockDim.y; comprueba que esta a la derecha de la columna con menor indice
        * j_seleccionada < blockIdx.y * blockDim.y + blockDim.y; comprueba que esta a la izquierda de la columna con mayor indice
        */
        if (i_seleccionada >= blockIdx.x * blockDim.x && i_seleccionada < blockIdx.x * blockDim.x + blockDim.x
            && j_seleccionada >= blockIdx.y * blockDim.y && j_seleccionada < blockIdx.y * blockDim.y + blockDim.y)
        {
            valor_pos_seleccionada = shared_tablero[(i_seleccionada%blockDim.x)*blockDim.y+(j_seleccionada%blockDim.y)];
        }
        else {
            valor_pos_seleccionada = dev_tablero[pos_seleccionada];
        }

        if (valor_pos_seleccionada == 8) { // seleccionado bloque bomba
            if (fila_o_columna == 0) // 0->fila, 1->columna
            {
                if (pos_hilo / dev_N == pos_seleccionada / dev_N) borrar = true;
            }
            else
            {
                if (pos_hilo % dev_N == pos_seleccionada % dev_N) borrar = true;
            }
        }
        else if (valor_pos_seleccionada == 9) { //seleccionado bloque TNT
            if (abs(pos_seleccionada / dev_N - pos_hilo / dev_N) <= 4 && abs(pos_seleccionada % dev_N - pos_hilo % dev_N) <= 4) {
                borrar = true;
            }
        }
        else if (valor_pos_seleccionada > 10) { //seleccionado bloque rompecabezas
            if (valor_propio_inicial == valor_pos_seleccionada % 10)
            {
                borrar = true;
            }
        }
        else { //seleccionado bloque normal
            borrar = esta_conectado(pos_hilo, pos_seleccionada, dev_tablero, dev_M, dev_N);
        }
        __syncthreads(); //fin buscar bloques que se van a eliminar
        if (borrar && pos_seleccionada != pos_hilo) { //el bloque seleccionado siempre tendr� borrar = true pero solo hay que borrarlo si hay alg�n otro bloque conectado as� que se har� despues
            dev_resultado[pos_hilo] = 0;
            atomicAdd(&dev_bloques_eliminados[0], 1);
        }
        else {
            dev_resultado[pos_hilo] = valor_propio_inicial;
        }
        __syncthreads(); //fin eliminar bloques normales
        //Colocar bloques especiales si hace falta
        if (pos_seleccionada == pos_hilo && valor_pos_seleccionada > 6) { // si se seleccion� un bloque especial no se va colocar otro aunque se hayan borrado 5 bloques o m�s
            atomicAdd(&dev_bloques_eliminados[0], 1);
            dev_resultado[pos_hilo] = 0;
        }
        else if (pos_seleccionada == pos_hilo && dev_bloques_eliminados[0] > 0) { // si se seleccion� un bloque normal colocamos bloques especiales en funci�n del n�mero de bloques eliminados en el turno
            atomicAdd(&dev_bloques_eliminados[0], 1);
            if (dev_bloques_eliminados[0] < 5) {
                dev_resultado[pos_hilo] = 0;
            }
            else if (dev_bloques_eliminados[0] == 5) {
                dev_resultado[pos_hilo] = 8; //bomba
            }
            else if (dev_bloques_eliminados[0] == 6) {
                dev_resultado[pos_hilo] = 9; //tnt
            }
            else if (dev_bloques_eliminados[0] >= 7) {
                hiprandState state;
                hiprand_init(seed, threadIdx.x, 0, &state);
                int color_rompecabezas = hiprand(&state) % colores + 1;
                dev_resultado[pos_hilo] = 10 + color_rompecabezas; //rompecabezas
            }
        }
        __syncthreads(); //fin colocar bloques especiales

        int valor_rellenar = rellenar(pos_hilo, dev_resultado, seed, colores, dev_M, dev_N);
        __syncthreads();
        dev_resultado[pos_hilo] = valor_rellenar;
        __syncthreads(); //fin rellenar

    }
}

void print_help() {
    printf("Usage: programa.exe [OPCIONES] DIFICULTAD FILAS COLUMNAS\n");
    printf("Dificultad = 1 -> 4 colores\nDificultad = 2 -> 6 colores\n");
    printf("Opciones:\n");
    printf("  -a,    Use automatic mode\n");
    printf("  -m,    Use difficult mode (default)\n");
}

int main(int argc, char* argv[])
{
    hipFree(0);
    srand(time(0));

    //gesti�n de los par�metros de ejecuci�n
    bool automatico = false;
    int dificultad = 2;
    int filas = 10, columnas = 10;
    for (int i = 1; i < argc; i++) { //ignoramos el primero (es el nombre del programa no nos interesa) y lo hacemos una vez por cada argumento
        if (strcmp(argv[i], "-a") == 0 || strcmp(argv[i], "-m") == 0) { //autom�tico o manual
            if (strcmp(argv[i], "-a") == 0) automatico = true;
        }
        else if (i == argc - 3 && (atoi(argv[i]) == 1 || atoi(argv[i]) == 2)) { // dificultad 1 o 2
            dificultad = atoi(argv[i]);
            if (dificultad == 1)
            {
                colores = 4;
            }
            else {
                colores = 6;
            }
        }
        else if (i == argc - 2) { //numero de filas
            filas = atoi(argv[i]);
        }
        else if (i == argc - 1) { // numero de columnas
            columnas = atoi(argv[i]);
        }
        else {
            printf("Invalid argument: %s\n", argv[i]);
            print_help();
            return 1;
        }
    }


    //Iniciamos el tablero
    int h_M = filas;
    int h_N = columnas;
    int* h_tablero = new int[h_M * h_N];
    for (int i = 0; i < h_M * h_N; i++) {
        h_tablero[i] = (rand() % colores) + 1;

    }

    //Lo mostramos

    printf("\n -- Tablero Inicial --\n");
    for (int x = 0; x < h_M * h_N; x++) {
        if (h_tablero[x] == 8)
        {
            printf("B  ");
        }
        else if (h_tablero[x] == 9) {
            printf("T  ");
        }
        else if (h_tablero[x] >= 10)
        {
            printf("R%d ", (h_tablero[x]) % 10);
        }
        else {
            printf("%d  ", h_tablero[x]);
        }
        if (x % h_N == h_N - 1) printf("\n");
    }

    while (vidas > 0)
    {
        //Pedimos al usuario que indique su movimiento
        int x_seleccionada;
        int y_seleccionada;
        bool es_entero;

        if (automatico)
        {
            printf("Haciendo jugada automatica\n");
            x_seleccionada = (rand() % h_M);
            y_seleccionada = (rand() % h_N);
            printf("Casilla: { %d, %d}\n", x_seleccionada, y_seleccionada);
        }
        else
        {
            do { //coordenada x
                cout << "Introduce x: ";
                cin >> x_seleccionada;

                es_entero = !cin.fail(); // No es un n�mero entero.

                if (!es_entero || x_seleccionada >= h_M) {
                    cin.clear(); // Limpia el error de cin.
                    cin.ignore(10000, '\n'); // Ignora todos los caracteres no v�lidos que se ingresaron.
                    cout << "Error: debes ingresar un numero entero entre 0 y " << h_M - 1 << endl;
                }
            } while (!es_entero || x_seleccionada >= h_M);

            do { //coordenada y
                cout << "Introduce y: ";
                cin >> y_seleccionada;

                es_entero = !cin.fail(); // No es un n�mero entero.

                if (!es_entero || y_seleccionada >= h_N) {
                    cin.clear(); // Limpia el error de cin.
                    cin.ignore(10000, '\n'); // Ignora todos los caracteres no v�lidos que se ingresaron.
                    cout << "Error: debes ingresar un numero entero entre 0 y " << h_N - 1 << endl;
                }
            } while (!es_entero || x_seleccionada >= h_N);
        }

        int h_pos[2] = { x_seleccionada, y_seleccionada };


        //Reservamos memoria para las matrices y las copiamos al Device
        int* h_resultado = new int[h_M * h_N];
        //int h_resultado[M][N];
        int* dev_tablero = new int[h_M * h_N];
        int* dev_resultado = new int[h_M * h_N];
        //int(*dev_tablero)[N], (*dev_resultado)[N];
        hipMalloc((void**)&dev_tablero, h_N * h_M * sizeof(int));
        hipMalloc((void**)&dev_resultado, h_N * h_M * sizeof(int));

        hipMemcpy(dev_tablero, h_tablero, h_M * h_N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_resultado, h_resultado, h_M * h_N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(dev_pos), h_pos, 2 * sizeof(int));


        int h_bloques_eliminados[1] = { 0 };
        int(*dev_bloques_eliminados)[1];
        hipMalloc((void**)&dev_bloques_eliminados, sizeof(int));
        hipMemcpy(dev_bloques_eliminados, h_bloques_eliminados, sizeof(int), hipMemcpyHostToDevice);

        int color_rompecabezas = (rand() % colores) + 1;

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        int MAX_HILOS = prop.maxThreadsPerBlock;
        // Mientras no se supere MAX_HILOS todo va bien
        int bloques_x = 2;
        int bloques_y = 2;
        int hilos_x = h_M / bloques_x;
        if (hilos_x * bloques_x < h_M) hilos_x++;
        int hilos_y = h_N / bloques_y;
        if (hilos_y * bloques_y < h_M) hilos_y++;
        int tam_shared = hilos_x* hilos_y * sizeof(int);

        //Iniciamos hilos y bloques
        dim3 blocksInGrid(bloques_x, bloques_y);
        dim3 threadsInBlock(hilos_x, hilos_y);
        addKernel << <blocksInGrid, threadsInBlock, tam_shared >> > (dev_tablero, dev_resultado, *dev_bloques_eliminados, (rand() % 2), (unsigned long long) time(NULL), colores, h_M, h_N);
        //rand() % 2 es un numero aleatorio entre 0 y 1 para saber si se borra la fila o la columna al encontrar una bomba, se pasa desde el host porque debe ser el mismo para todos los hilos

        hipMemcpy(h_bloques_eliminados, dev_bloques_eliminados, sizeof(int), hipMemcpyDeviceToHost);
        //Copiamos el resultado al host
        hipMemcpy(h_resultado, dev_resultado, h_M * h_N * sizeof(int), hipMemcpyDeviceToHost);

        //Y lo mostramos
        for (int x = 0; x < h_M * h_N; x++) {
            if (h_resultado[x] == 8)
            {
                printf("B  ");
            }
            else if (h_resultado[x] == 9)
            {
                printf("T  ");
            }
            else if (h_resultado[x] >= 10)
            {
                printf("R%d ", (h_resultado[x]) % 10);
            }
            else
            {
                printf("%d  ", h_resultado[x]);
            }
            if (x % h_N == h_N - 1) printf("\n");
        }

        printf("Bloques eliminados en este movimiento: %d\n", h_bloques_eliminados[0]);
        if (h_bloques_eliminados[0] == 0) {
            vidas--;
            printf("Has perdido una vida, te quedan %d\n", vidas);
        }

        for (int i = 0; i < h_M * h_N; i++) h_tablero[i] = h_resultado[i];

        //Por ultimo liberamos memoria
        hipFree(dev_tablero); hipFree(dev_resultado); hipFree(dev_pos); hipFree(dev_bloques_eliminados);
    }

    printf("Has perdido");

    return 0;
}